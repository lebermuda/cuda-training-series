#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

template <typename T>
void alloc_bytes(T &ptr, size_t num_bytes){

  ptr = (T)malloc(num_bytes);
}

__global__ void inc(int *array, size_t n){
  size_t idx = threadIdx.x+blockDim.x*blockIdx.x;
  while (idx < n){
    array[idx]++;
    idx += blockDim.x*gridDim.x; // grid-stride loop
    }
}

__global__ void kernel_changeChar(char* data,int index, char caractere){
    data[index]=caractere;
}

const size_t  ds = 32ULL*1024ULL*1024ULL;

int main(){
    char* data;
    hipMallocManaged(&data,2*sizeof(char));

    kernel_changeChar<<<1,1>>>(data,1,"k");
    data[0]="O";

    //hipDeviceSynchronize();
    //printf("%d\n",data);

    hipFree(data);

//    int *h_array;
//    hipMallocManaged(&h_array, ds*sizeof(d_array[0]));
//
//    memset(h_array, 0, ds*sizeof(h_array[0]));
//
//    inc<<<256, 256>>>(h_array, ds);
//    hipDeviceSynchronize();
//
//    for (int i = 0; i < ds; i++)
//        if (h_array[i] != 1) {printf("mismatch at %d, was: %d, expected: %d\n", i, h_array[i], 1); hipFree(h_array); return -1;}
//    printf("success!\n");
//    hipFree(h_array);
//    return 0;

//  int *h_array, *d_array;
//  alloc_bytes(h_array, ds*sizeof(h_array[0]));
//  hipMalloc(&d_array, ds*sizeof(d_array[0]));
//  //cudaCheckErrors("hipMalloc Error");
//  memset(h_array, 0, ds*sizeof(h_array[0]));
//  hipMemcpy(d_array, h_array, ds*sizeof(h_array[0]), hipMemcpyHostToDevice);
//  //cudaCheckErrors("hipMemcpy H->D Error");
//  inc<<<256, 256>>>(d_array, ds);
//  //cudaCheckErrors("kernel launch error");
//  hipMemcpy(h_array, d_array, ds*sizeof(h_array[0]), hipMemcpyDeviceToHost);
//  //cudaCheckErrors("kernel execution or hipMemcpy D->H Error");
//  for (int i = 0; i < ds; i++)
//    if (h_array[i] != 1) {printf("mismatch at %d, was: %d, expected: %d\n", i, h_array[i], 1); return -1;}
//  printf("success!\n");
//  return 0;
}
