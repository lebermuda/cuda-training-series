#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>


struct list_elem {
    int key;
    list_elem *next;
};



__global__ void gpu_kernel(double* matrix,int n){
    int i =threadIdx.x+blockDim.x*blockIdx.x;
    int j= threadIdx.y+blockDim.y*blockIdx.y;
    int index = i*n+j;
    matrix[index]=index;
}

void printMatrix(double* matrix){
    int index;
    for (int i = 0;i<num_elem;i++){
        for (int j = 0;j<num_elem;j++){
            index=i*num_elem+j;
            printf("%f ",matrix[index]);
        }
        printf("\n");
    }
}

const int num_elem = 5;
int main(){

    double* matrix ;

    hipMallocManaged(&matrix,num_elem*num_elem*sizeof(double));


    gpu_kernel<<<5,5>>>(matrix);
    hipDeviceSynchronize();

    printMatrix(matrix);

    free(matrix);

}
