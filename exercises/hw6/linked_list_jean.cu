#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>


struct list_elem {
    int key;
    list_elem *next;
};



__global__ void gpu_kernel(double* matrix,int n){
    int i =threadIdx.x+blockDim.x*blockIdx.x;
    int j= threadIdx.y+blockDim.y*blockIdx.y;
    int index = i*n+j;
    matrix[index]=index;
}

void printMatrix(double* matrix,int n){
    int index;
    for (int i = 0;i<n;i++){
        for (int j = 0;j<n;j++){
            index=i*n+j;
            printf("%f ",matrix[index]);
        }
        printf("\n");
    }
}

const int num_elem = 5;
int main(){

    double* matrix ;

    hipMallocManaged(&matrix,num_elem*num_elem*sizeof(double));

    dim3 dimGrid (1,1,1);
    dim3 dimBlock (num_elem,num_elem-2,1);
    gpu_kernel<<<dimGrid,dimBlock>>>(matrix,num_elem);
    hipDeviceSynchronize();

    printMatrix(matrix,num_elem);

    free(matrix);

}
