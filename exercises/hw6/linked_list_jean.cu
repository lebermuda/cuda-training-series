#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>


struct list_elem {
    int key;
    list_elem *next;
};

template <typename T>
void alloc_bytes(T &ptr, size_t num_bytes){

    //ptr = (T)malloc(num_bytes);
    hipMallocManaged(&ptr,num_bytes);
}

__host__ __device__
void print_element(list_elem *list, int ele_num) {
    list_elem *elem = list;
    for (int i = 0; i < ele_num; i++) {
        elem = elem->next;
        printf("key = %d\n", elem->key);
    }
}

__global__ void gpu_print_element(list_elem *list, int ele_num){
    print_element(list, ele_num);
}
const int ele = 3;



__global__ void gpu_kernel(double* matrix,int n){
    int i =threadIdx.x+blockDim.x*blockIdx.x;
    int j= threadIdx.y+blockDim.y*blockIdx.y;
    int index = i*n+j;
    matrix[index]=index;
}

void printMatrix(double* matrix,int n){
    int index;
    for (int i = 0;i<n;i++){
        for (int j = 0;j<n;j++){
            index=i*n+j;
            printf("%f ",matrix[index]);
        }
        printf("\n");
    }
}

void addElement(double* matrix, int n, int i_start,int j_start){
    int index;
    for (int i = i_start;i<n;i++){
        for (int j = j_start;j<n;j++){
            index=i*n+j;
            matrix[index]=1;
        }
    }
}

const int num_elem = 5;
int main(){

//    double* matrix ;
//
//    hipMallocManaged(&matrix,num_elem*num_elem*sizeof(double));
//
//    dim3 dimGrid (1,1,1);
//    dim3 dimBlock (num_elem-2,num_elem,1);
//    gpu_kernel<<<dimGrid,dimBlock>>>(matrix,num_elem);
//
//    //MARCHE PAS EN PARALLEL
//    //addElement(matrix,num_elem,num_elem-2,0);
//
//    hipDeviceSynchronize();
//
//    //MARCHE EN SEQUENTIEL
//    //addElement(matrix,num_elem,num_elem-2,0);
//
//
//    printMatrix(matrix,num_elem);
//
//    hipFree(matrix);

    list_elem *list_base, *list;
    alloc_bytes(list_base, sizeof(list_elem));
    list = list_base;
    for (int i = 0; i < num_elem; i++){
        list->key = i;
        alloc_bytes(list->next, sizeof(list_elem));
        list = list->next;}
    print_element(list_base, ele);
    gpu_print_element<<<1,1>>>(list_base, ele);
    hipDeviceSynchronize();

    hipFree(list_base);
}
